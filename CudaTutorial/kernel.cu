﻿#include "hip/hip_runtime.h"

#include <stdlib.h>     
#include <iostream>
#include <algorithm>

__global__ 
void matmul(int* a, int* b, int* c, int N)
{
	int temp = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < N && col < N)
	{
		for (int k = 0; k < N; ++k)
		{
			temp += a[row * N + k] + b[k * N + col];
		}
		c[row * N + col] = temp;
	}
}

void verify_mat_mul(int* a, int* b, int* c, int N)
{
	int temp = 0;
	int max_err = 0;
	for (int row = 0; row < N; ++row)
	{
		for (int col = 0; col < N; ++col)
		{
			for (int k = 0; k < N; ++k)
			{
				temp += a[row * N + k] + b[k * N + col];
			}
			max_err = std::max(0, temp);
		}
	}
	std::cout << "MAX ERROR: " << max_err << std::endl;
}




int main()
{
	int n = 1 << 10;		//2^16

	int bytes = n * n * sizeof(int);
	int* a_host = new int[n * n];
	int* b_host = new int[n * n];
	int* c_host = new int[n * n];

	int* a_device;
	int* b_device;
	int* c_device;

	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{	
			a_host[i * n + j] = rand() % 100;
			b_host[i * n + j] = rand() % 100;
		}
	}

	hipMalloc(&a_device, bytes);
	hipMalloc(&b_device, bytes);
	hipMalloc(&c_device, bytes);
	hipMemcpy(a_device, a_host, bytes, hipMemcpyHostToDevice);
	hipMemcpy(b_device, b_host, bytes, hipMemcpyHostToDevice);

	const int numthreads = 32;
	int numblocks = (int)ceil(n / numthreads);

	dim3 blocks(numblocks, numblocks);		//grid size
	dim3 threads(numthreads, numthreads);	//thread per block

	matmul<<<blocks, threads>>> (a_device, b_device, c_device, n);

	hipDeviceSynchronize();

	hipMemcpy(c_host, c_device, bytes, hipMemcpyDeviceToHost);
	
	verify_mat_mul(a_host, c_host, c_host, n);

	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);
}
